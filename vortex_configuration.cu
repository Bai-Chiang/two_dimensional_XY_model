#include "hip/hip_runtime.h"
// --- file: vortex_configuration.cu ---
// this file calculate a 2D XY-model with CUDA
// get one 2D-spin lattice result in specific temperature
// output spin direction:theta in directory result/vortex_configuration/




#include <cstdio>
#include <iostream>
#include <cstdio>
#include <cmath>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <string>
#include "src/xy_model.h"


int main() {

    // some values can be adjusted
    int length = 128; // 2^n, n >= 5    length of 2D-spins, the 2D-spins lattice will be length * length
    unsigned int seed = 0; // seed of random numbers    set seed to a fix number, so that each time you run will get same result
    long long warm_up_steps = length * length * 65536; // length * length * 2^n    warm up step is proportional to total number of spins
    float T = 0.001; // temperature, suppose boltzmann constant k = 1 



    // ============================================================================================
    long long n_sample = 1; // plot one figure of 2D-spins, hence n_sample = 1
    long long size = length * length; // the total size of 2D-spin lattice is length * length
    int threads_per_block = std::min(1024LL, size);
    int blocks = size/threads_per_block;
    long long n_itter = warm_up_steps/size; // sice total number of threads is size (threads_per_block * blocks), to obtain warm_up_steps, need to itterate warm_up_steps/size times
    
    // --------------------------------------------------------------------------------------------
    hiprandState_t* states; // used to store random state for each core
    hipMalloc((void**) &states, size * sizeof(hiprandState_t)); // allocate memory in device
    init_rand<<<blocks, threads_per_block>>>(seed, states); // initialize for all states


    // allocate memory in device
    float* d_spins; // device copy of a spin system (pp_spins[i])
    int* d_length; // device copy of length
    long long* d_n_itter; // device copy of n_itter
    float* d_T; // device copy of T
    hipMalloc((void**)&d_spins, size * sizeof(float));
    hipMalloc((void**)&d_length, sizeof(int));
    hipMalloc((void**)&d_n_itter, sizeof(long long));
    hipMalloc((void**)&d_T, sizeof(float));
    hipMemcpy(d_length, &length, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_n_itter, &n_itter, sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(d_T, &T, sizeof(float), hipMemcpyHostToDevice);

    float* p_spins = new float[size];

    for (long long i = 0; i < n_sample; ++i) {

        // initialize spins
        initialize <<<blocks, threads_per_block>>> (d_spins, states);

        // warm up
        warm_up <<<blocks, threads_per_block>>> (d_spins, d_T, d_length, d_n_itter, states);

    }

    // copy memory from device to host
    hipMemcpy(p_spins, d_spins, size * sizeof(float), hipMemcpyDeviceToHost);

    // --------------------------------------------------------------------------------------------
    // write file

    std::string str_L = std::to_string(length);
    std::string str_T = std::to_string(T);
    std::string str_n_warm = std::to_string(warm_up_steps);
    std::string file_name = std::string("result/vortex_configuration/") + str_L + std::string("_") + str_T + "_" + str_n_warm + std::string(".data");
    
    FILE* pfile;
    pfile = fopen(file_name.c_str(), "w");
    if (pfile != NULL) {
        fprintf(pfile, "%d\n%f\n%lld\n", length, T, warm_up_steps);
        for (int i = 0; i < length; ++i) {
            for (int j = 0; j < length; ++j) {
                fprintf(pfile, "%f,", p_spins[i*length + j]);
            }
            fprintf(pfile, "\n");
        }
    } else {
        printf("ERROR, unable to open file 'result/spins.csv' !");
    }
    fclose(pfile);

    // -----------------------------------------------------------------
    hipFree(&d_length); hipFree(d_spins);
    hipFree(states);
    delete[] p_spins;

    return 0;
}


