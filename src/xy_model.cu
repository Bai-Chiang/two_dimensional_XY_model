#include "hip/hip_runtime.h"
#include <cmath>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <cstdio>

#include "xy_model.h"


/* ------------------------------------------------------------------------------------------------
 * initize seed and sequence for all device cores
 * all cores will have same seed, but different sequence number
 * so each core will generate different random number series
 */
__global__ void init_rand(unsigned int seed, hiprandState_t* states) {
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    hiprand_init(
            seed, // seed value is same for each core
            index, // sequence number for each core is it's index
            0, // offset is 0
            &states [index]
            );
}



/* ------------------------------------------------------------------------------------------------
 * initialize the spins with random direction from [0, 2*pi]
 */
__global__ void initialize (float* spins, hiprandState_t* states) {
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    spins[index] = hiprand_uniform(&states[index]) * 2*M_PI;
}




/* ------------------------------------------------------------------------------------------------
 * warm up the system
 * n_itter is the itteration times for each thread
 * the total warm_up_step for the whole system is n_itter * n_block * n_threads_per_block
 */
__global__ void warm_up (float* spins, float* p_T, int* p_length, long long* p_n_itter, hiprandState_t* states) {
    long long n_itter = *p_n_itter;
    int length = *p_length;
    float T = *p_T;
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int dim1, dim2;
    float upper_spin, lower_spin, left_spin, right_spin;
    float current_energy, changed_energy, delta_energy;
    float new_spin, current_spin;
    for (long long i = 0; i < n_itter; ++i){
        dim1 = hiprand(&states[index])%length;
        dim2 = hiprand(&states[index])%length;

        // get current position spin
        current_spin = spins[dim1 * length + dim2];

        // get upper spin
        if (dim1 != 0) {
            upper_spin = spins[(dim1-1) * length + dim2];
        } else {
            upper_spin = spins[(length-1) * length + dim2];
        }

        // get lower spin
        if (dim1 != length-1) {
            lower_spin = spins[(dim1+1) * length + dim2];
        } else {
            lower_spin = spins[0 * length + dim2];
        }

        // get left spin
        if (dim2 != 0) {
            left_spin = spins[dim1 * length + (dim2-1)];
        } else {
            left_spin = spins[dim1 * length + (length-1)];
        }

        // get right spin
        if (dim2 != length-1) {
            right_spin = spins[dim1 * length + (dim2+1)];
        } else {
            right_spin = spins[dim1 * length + 0];
        }

        // get energy
        current_energy = -(std::cos(current_spin - upper_spin) + std::cos(current_spin - lower_spin)
                           + std::cos(current_spin - left_spin) + std::cos(current_spin - right_spin));

        // new spin direction
        new_spin = hiprand_uniform(&states[index]) * 2*M_PI;

        // changed energy
        changed_energy = -(std::cos(new_spin - upper_spin) + std::cos(new_spin - lower_spin)
                           + std::cos(new_spin - left_spin) + std::cos(new_spin - right_spin));

        // decide whether change spin
        delta_energy = changed_energy - current_energy;
        if ( hiprand_uniform(&states[index]) < std::exp(-delta_energy/T) ) {
            spins[dim1 * length + dim2] = new_spin;
        }

    }
}



/* ------------------------------------------------------------------------------------------------
 * get x and y component of spins
 */
__global__ void get_spin(float* Sx, float* Sy, float* theta) {
    long long global_id = blockIdx.x *blockDim.x + threadIdx.x;
    Sx[global_id] = std::cos(theta[global_id]);
    Sy[global_id] = std::sin(theta[global_id]);
}


/* ------------------------------------------------------------------------------------------------
 * get system energy for each spin
 */
__global__ void get_energy (float* energy, float* spins, int* p_length) {
    int length = *p_length;
    long long global_id = blockIdx.x * blockDim.x + threadIdx.x;
    float current_spin, upper_spin, lower_spin, left_spin, right_spin;

    // get current position spin
    current_spin = spins[blockIdx.x * length + threadIdx.x];

    // get upper spin
    if (blockIdx.x == 0) {
        upper_spin = spins[(length-1) * length + threadIdx.x];
    } else {
        upper_spin = spins[(blockIdx.x-1) * length + threadIdx.x];
    }

    // get lower spin
    if (blockIdx.x == length) {
        lower_spin = spins[0 * length + threadIdx.x];
    } else {
        lower_spin = spins[(blockIdx.x+1) * length + threadIdx.x];
    }

    // get left spin
    if (threadIdx.x != 0) {
        left_spin = spins[blockIdx.x * length + (threadIdx.x-1)];
    } else {
        left_spin = spins[blockIdx.x * length + (length-1)];
    }

    // get right spin
    if (threadIdx.x != length-1) {
        right_spin = spins[blockIdx.x * length + (threadIdx.x+1)];
    } else {
        right_spin = spins[blockIdx.x * length + 0];
    }

    // get energy
    energy[global_id] = -(std::cos(current_spin - upper_spin) + std::cos(current_spin - lower_spin)
                          + std::cos(current_spin - left_spin) + std::cos(current_spin - right_spin));
}

/* ------------------------------------------------------------------------------------------------
 * warm up the system
 * warm up type 1 will warm up these kind of points. 1 means warm up, 0 means not warm up.
 * each 1 will be warm up one time
 *   1 0 1 0 1 0 1 0 ...
 *   0 1 0 1 0 1 0 1 ...
 *   1 0 1 0 1 0 1 0 ...
 *   0 1 0 1 0 1 0 1 ...
 *   1 0 1 0 1 0 1 0 ...
 *   0 1 0 1 0 1 0 1 ...
 *   1 0 1 0 1 0 1 0 ...
 *   0 1 0 1 0 1 0 1 ...
 *   . . . . . . . . .
 *   . . . . . . . .  .
 *   . . . . . . . .    .
 */
__global__ void warm_up_type_1 (float* spins, float* p_T, int* p_length, hiprandState_t* states) {
    int length = *p_length;
    float T = *p_T;
    int dim1 = blockIdx.x;
    int dim2 = blockIdx.x%2 + (threadIdx.x * 2);
    long long global_id = blockIdx.x * (2LL*blockDim.x) + blockIdx.x%2 + (threadIdx.x * 2LL);
    float upper_spin, lower_spin, left_spin, right_spin;
    float current_energy, changed_energy, delta_energy;
    float new_spin, current_spin;
/*
    // printf("%f\n", spins[16383]);
    if (global_id == 16383) 
        printf("%f", spins[(dim1-1) * length + dim2]);
*/    

    // get current position spin
    current_spin = spins[global_id];

    // get upper spin
    if (dim1 != 0) {
        upper_spin = spins[(dim1-1) * length + dim2];
    } else {
        upper_spin = spins[(length-1) * length + dim2];
    }

    // get lower spin
    if (dim1 != length-1) {
        lower_spin = spins[(dim1+1) * length + dim2];
    } else {
        lower_spin = spins[0 * length + dim2];
    }

    // get left spin
    if (dim2 != 0) {
        left_spin = spins[dim1 * length + (dim2-1)];
    } else {
        left_spin = spins[dim1 * length + (length-1)];
    }

    // get right spin
    if (dim2 != length-1) {
        right_spin = spins[dim1 * length + (dim2+1)];
    } else {
        right_spin = spins[dim1 * length + 0];
    }

    // get energy
    current_energy = -(std::cos(current_spin - upper_spin) + std::cos(current_spin - lower_spin)
                       + std::cos(current_spin - left_spin) + std::cos(current_spin - right_spin));

    // new spin direction
    new_spin = hiprand_uniform(&states[global_id]) * 2*M_PI;

    // changed energy
    changed_energy = -(std::cos(new_spin - upper_spin) + std::cos(new_spin - lower_spin)
                       + std::cos(new_spin - left_spin) + std::cos(new_spin - right_spin));
 
    // decide whether change spin
    delta_energy = changed_energy - current_energy;
    if ( hiprand_uniform(&states[global_id]) < std::exp(-delta_energy/T) ) {
        spins[global_id] = new_spin;
    }

}


/* ------------------------------------------------------------------------------------------------
 * warm up the system
 * warm up type 2 will warm up these kind of points. 1 means warm up, 0 means not warm up.
 * each 1 will be warm up one time
 *   0 1 0 1 0 1 0 1 ...
 *   1 0 1 0 1 0 1 0 ...
 *   0 1 0 1 0 1 0 1 ...
 *   1 0 1 0 1 0 1 0 ...
 *   0 1 0 1 0 1 0 1 ...
 *   1 0 1 0 1 0 1 0 ...
 *   0 1 0 1 0 1 0 1 ...
 *   1 0 1 0 1 0 1 0 ...
 *   . . . . . . . . .
 *   . . . . . . . .  .
 *   . . . . . . . .    .
 */
__global__ void warm_up_type_2 (float* spins, float* p_T, int* p_length, hiprandState_t* states) {
    int length = *p_length;
    float T = *p_T;
    int dim1 = blockIdx.x;
    int dim2 = (1-blockIdx.x%2) + (threadIdx.x * 2);
    long long global_id = blockIdx.x * (2*blockDim.x) + (1-blockIdx.x%2) + (threadIdx.x * 2);
    float upper_spin, lower_spin, left_spin, right_spin;
    float current_energy, changed_energy, delta_energy;
    float new_spin, current_spin;

    // get current position spin
    current_spin = spins[global_id];

    // get upper spin
    if (dim1 != 0) {
        upper_spin = spins[(dim1-1) * length + dim2];
    } else {
        upper_spin = spins[(length-1) * length + dim2];
    }

    // get lower spin
    if (dim1 != length-1) {
        lower_spin = spins[(dim1+1) * length + dim2];
    } else {
        lower_spin = spins[0 * length + dim2];
    }

    // get left spin
    if (dim2 != 0) {
        left_spin = spins[dim1 * length + (dim2-1)];
    } else {
        left_spin = spins[dim1 * length + (length-1)];
    }

    // get right spin
    if (dim2 != length-1) {
        right_spin = spins[dim1 * length + (dim2+1)];
    } else {
        right_spin = spins[dim1 * length + 0];
    }

    // get energy
    current_energy = -(std::cos(current_spin - upper_spin) + std::cos(current_spin - lower_spin)
                       + std::cos(current_spin - left_spin) + std::cos(current_spin - right_spin));

    // new spin direction
    new_spin = hiprand_uniform(&states[global_id]) * 2*M_PI;

    // changed energy
    changed_energy = -(std::cos(new_spin - upper_spin) + std::cos(new_spin - lower_spin)
                       + std::cos(new_spin - left_spin) + std::cos(new_spin - right_spin));

    // decide whether change spin
    delta_energy = changed_energy - current_energy;
    if ( hiprand_uniform(&states[global_id]) < std::exp(-delta_energy/T) ) {
        spins[global_id] = new_spin;
    }
}


