//
// Created by Baiqiang Qiang on 30/05/2017.
//


#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <cmath>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

/* ------------------------------------------------------------------------------------------------
 * initial seed and sequence for all device cores
 * all cores will have same seed, but different sequence number
 * so each core will generate different random number series 
 */
__global__ void init_rand(unsigned int seed, hiprandState_t* states) {
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    hiprand_init(
            seed, // seed value is same for each core
            index, // sequence number for each core is it's index
            0, // offset is 0
            &states [index]
            );
}

/* ------------------------------------------------------------------------------------------------
 * initialize the spins with random direction from [0, 2*pi]
 */
__global__ void initialize (float* d_spins, hiprandState_t* states) {
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    d_spins[index] = hiprand_uniform(&states[index]) * 2*M_PI;
}

void print_spins (float* spins, int length, int print_dim1, int print_dim2) {
    for (int i = 0; i < print_dim1; ++i) {
        for (int j = 0; j < print_dim2; ++j) {
            printf("%f ", spins[i*length + j]);
            //std::cout << spins[i*length + j] << " ";
        }
        //std::cout << std::endl;
        printf("\n");
    }
    //std::cout << std::endl;
    printf("\n");
}

// ================================================================================================

int main() {
    long long n_sample = 10;
    long long warm_up_steps = 100;
    int length = 512; // 2^n, n >= 5 
    unsigned int seed = 0;
    long long size = length * length;
    int threads_per_block = 1024;
    int blocks = size/threads_per_block;

    // --------------------------------------------------------------------------------------------
    hiprandState_t* states; // used to store random state for each core
    hipMalloc((void**) &states, size * sizeof(hiprandState_t)); // allocate memory in device
    init_rand<<<blocks, threads_per_block>>>(seed, states); // initialize for all states
    
    //std::cout << "init_rand completed" << std::endl;
    
    // allocate memory in device
    float* d_spins; // device copy of a spin system (pp_spins[i])
    int* d_length; // device copy of length
    hipMalloc((void**)&d_spins, size * sizeof(float));
    hipMalloc((void**)&d_length, sizeof(int));
    hipMemcpy(d_length, &length, sizeof(int), hipMemcpyHostToDevice);

    float* p_spins = new float[size];
    hipDeviceSynchronize();

    
    for (long long i = 0; i < n_sample; ++i) {
       // initialize spins
        initialize<<<blocks, threads_per_block>>>(d_spins, states);

        // copy memory from device to host
        hipMemcpy(p_spins, d_spins, size * sizeof(float), hipMemcpyDeviceToHost);
        
        std::cout << i << std::endl;
        hipDeviceSynchronize();

    }
    
    print_spins (p_spins, length, 10, 15);

    // -----------------------------------------------------------------
    hipFree(&d_length); hipFree(d_spins);
    hipFree(states); 
    delete[] p_spins;
    return 0;
}
